#include "hip/hip_runtime.h"
%%cu
#include <iostream>
using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 4;
    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}





%%cu
#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

// CUDA kernel for vector addition
__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    // Size of the vectors
    int N = 1000000;

    // Host vectors
    int* A = new int[N];
    int* B = new int[N];
    int* C = new int[N];

    // Initialize host vectors with random values
    for (int i = 0; i < N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // Device vectors
    int* X, * Y, * Z;
    hipMalloc(&X, N * sizeof(int));
    hipMalloc(&Y, N * sizeof(int));
    hipMalloc(&Z, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(X, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Y, B, N * sizeof(int), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Measure execution time of parallel algorithm
    auto startParallel = high_resolution_clock::now();
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    hipDeviceSynchronize();
    auto stopParallel = high_resolution_clock::now();
    auto durationParallel = duration_cast<milliseconds>(stopParallel - startParallel);
    cout << "Parallel Execution Time: " << durationParallel.count() << " milliseconds" << endl;

    // Free device memory
    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    // Free host memory
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}





sequential

#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

// Sequential implementation of vector addition
void addSequential(int* A, int* B, int* C, int size) {
    for (int i = 0; i < size; i++) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    // Size of the vectors
    int N = 1000000;

    // Dynamic memory allocation for arrays A, B, and C
    int* A = new int[N];
    int* B = new int[N];
    int* C = new int[N];

    // Initialize vectors A and B with random values
    for (int i = 0; i < N; i++) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // Measure execution time of sequential algorithm
    auto startSequential = high_resolution_clock::now();
    addSequential(A, B, C, N);
    auto stopSequential = high_resolution_clock::now();
    auto durationSequential = duration_cast<milliseconds>(stopSequential - startSequential);
    cout << "Sequential Execution Time: " << durationSequential.count() << " milliseconds" << endl;

    // Free allocated memory
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}



!pip install git+https://github.com/afnan47/cuda.git
%load_ext nvcc_plugin

%%cu